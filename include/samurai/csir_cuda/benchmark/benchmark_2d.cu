#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <cstring>

// Original CPU implementation for comparison
#include "../../csir_unified/src/csir.hpp"
// Our new CUDA implementation
#include "../src/csir_cuda.cuh"

// Use a namespace alias to distinguish between CPU and CUDA versions
namespace csir_cpu = csir;

// Data generation function (adapted to produce both CPU and CUDA host types)
namespace {

csir_cpu::CSIR_Level create_fragmented_mesh_cpu(int size, float density, std::size_t level, unsigned seed) {
    csir_cpu::CSIR_Level mesh;
    mesh.level = level;
    mesh.intervals_ptr.push_back(0);

    std::mt19937 gen(seed);
    std::uniform_real_distribution<> dis(0.0, 1.0);

    for (int y = 0; y < size; ++y) {
        bool row_has_intervals = false;
        int last_x = -1;
        for (int x = 0; x < size; ++x) {
            if (dis(gen) < density) {
                if (last_x == -1) last_x = x;
            } else {
                if (last_x != -1) {
                    mesh.intervals.push_back({last_x, x});
                    last_x = -1;
                    row_has_intervals = true;
                }
            }
        }
        if (last_x != -1) {
            mesh.intervals.push_back({last_x, size});
            row_has_intervals = true;
        }
        if(row_has_intervals) {
            mesh.y_coords.push_back(y);
            mesh.intervals_ptr.push_back(mesh.intervals.size());
        }
    }
    return mesh;
}

// Helper to convert from CPU struct to CUDA host struct (they are identical in layout)
csir::cuda::CSIR_Level_Host convert_to_cuda_host(const csir_cpu::CSIR_Level& cpu_level) {
    csir::cuda::CSIR_Level_Host host_level;
    host_level.level = cpu_level.level;
    host_level.y_coords = cpu_level.y_coords;
    host_level.intervals_ptr = cpu_level.intervals_ptr;
    // The interval struct is also identical, so we can just copy
    host_level.intervals.resize(cpu_level.intervals.size());
    std::memcpy(host_level.intervals.data(), cpu_level.intervals.data(), cpu_level.intervals.size() * sizeof(csir_cpu::Interval));
    return host_level;
}

}

int main() {
    // --- Benchmark Parameters ---
    const int size = 4096; // Large mesh size
    const float density = 0.1f; // Sparsity of the mesh
    const unsigned seed1 = 1337;
    const unsigned seed2 = 4242;

    std::cout << "--- CSIR 2D CPU vs CUDA Performance Benchmark ---" << std::endl;
    std::cout << "Mesh size: " << size << "x" << size << ", Density: " << density << std::endl << std::endl;

    // --- Data Generation ---
    auto A_cpu = create_fragmented_mesh_cpu(size, density, 5, seed1);
    auto B_cpu = create_fragmented_mesh_cpu(size, density, 5, seed2);

    auto A_cuda_host = convert_to_cuda_host(A_cpu);
    auto B_cuda_host = convert_to_cuda_host(B_cpu);

    // --- CPU Timings ---
    double cpu_union_time, cpu_intersection_time;

    {
        auto start = std::chrono::high_resolution_clock::now();
        auto res_union_cpu = csir_cpu::union_(A_cpu, B_cpu);
        auto end = std::chrono::high_resolution_clock::now();
        cpu_union_time = std::chrono::duration<double, std::milli>(end - start).count();
    }
    {
        auto start = std::chrono::high_resolution_clock::now();
        auto res_intersect_cpu = csir_cpu::intersection(A_cpu, B_cpu);
        auto end = std::chrono::high_resolution_clock::now();
        cpu_intersection_time = std::chrono::duration<double, std::milli>(end - start).count();
    }

    // --- GPU Timings ---
    float gpu_union_time, gpu_intersection_time;
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    {
        hipEventRecord(start_event);
        auto res_union_gpu = csir::cuda::union_(A_cuda_host, B_cuda_host);
        hipEventRecord(stop_event);
        hipEventSynchronize(stop_event);
        hipEventElapsedTime(&gpu_union_time, start_event, stop_event);
    }
    {
        hipEventRecord(start_event);
        auto res_intersect_gpu = csir::cuda::intersection(A_cuda_host, B_cuda_host);
        hipEventRecord(stop_event);
        hipEventSynchronize(stop_event);
        hipEventElapsedTime(&gpu_intersection_time, start_event, stop_event);
    }

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // --- Results ---
    std::cout << std::fixed << std::setprecision(4);
    std::cout << "+--------------+------------------+------------------+" << std::endl;
    std::cout << "| Operation    | CPU Time (ms)    | GPU Time (ms)    |" << std::endl;
    std::cout << "+--------------+------------------+------------------+" << std::endl;
    std::cout << "| Union        | " << std::setw(16) << cpu_union_time << " | " << std::setw(16) << gpu_union_time << " |" << std::endl;
    std::cout << "| Intersection | " << std::setw(16) << cpu_intersection_time << " | " << std::setw(16) << gpu_intersection_time << " |" << std::endl;
    std::cout << "+--------------+------------------+------------------+" << std::endl;

    return 0;
}